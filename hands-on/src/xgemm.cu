#include "hip/hip_runtime.h"
#include <stdio.h> // needed for ‘printf’ 
#include <stdlib.h> // needed for ‘RAND_MAX’ 
#include <omp.h> // needed for OpenMP 
#include <time.h> // needed for clock() and CLOCKS_PER_SEC etc
#include "helper.h" // local helper header to clean up code

#ifdef USE_DOUBLE
typedef double X_TYPE;
#else
typedef float X_TYPE;
#endif

void initialize_matrices(X_TYPE* A, X_TYPE* B, X_TYPE* C, int ROWS, int COLUMNS){
    for (int i = 0; i < ROWS * COLUMNS; i++)
        {
            A[i] = (X_TYPE) rand() / RAND_MAX ;
            B[i] = (X_TYPE) rand() / RAND_MAX ;
            C[i] = 0.0 ;
        }
}
__global__ void simple_matrix_multiply(X_TYPE* D_A, X_TYPE* D_B, X_TYPE* D_C, int ROWS, int COLUMNS){
    
    int local_COLUMN = threadIdx.x + blockIdx.x * blockDim.x;
		int local_ROW = threadIdx.y + blockIdx.y * blockDim.y;
		int local_index = local_COLUMN + local_ROW * ROWS; // Right now this only works for symetric matricies
		int tmp = 0;  
    
    if(local_ROW < ROWS && local_COLUMN < COLUMNS){
			for(int k=0; k<COLUMNS; k++){
				tmp += D_A[local_ROW * ROWS + k] * D_B[k * COLUMNS + local_COLUMN];
			}
			D_C[local_index] = tmp;
		}
}


int main( int argc, char *argv[] )  {

  int ROWS;
  int COLUMNS;
  int N;

  /* DUMB bools needed for the argument parsing logic */
  bool openmp = false;
  bool simple = true;
  bool sanity_check = false;
  
  /* VERY DUMB Argument Parsers */
  N = parse_arguments(argc, argv, &simple, &openmp, &sanity_check);
  ROWS = N;
  COLUMNS = N;


  /* declare the arrays...  better to do it as 1D arrays for CUDA */

  // First allocated them on the host (CPU)
    X_TYPE* A = (X_TYPE*)malloc((ROWS * COLUMNS) * sizeof(X_TYPE));
    X_TYPE* B = (X_TYPE*)malloc((ROWS * COLUMNS) * sizeof(X_TYPE));
    X_TYPE* C = (X_TYPE*)malloc((ROWS * COLUMNS) * sizeof(X_TYPE));

  // Then Allocate them on the GPUs
  X_TYPE* D_A;
  X_TYPE* D_B;
  X_TYPE* D_C;
  hipMalloc((void**)&D_A, sizeof( X_TYPE ) * (ROWS * COLUMNS));
  hipMalloc((void**)&D_B, sizeof( X_TYPE ) * (ROWS * COLUMNS));
  hipMalloc((void**)&D_C, sizeof( X_TYPE ) * (ROWS * COLUMNS));

  /* initialize the arrays */
  clock_t t; // declare clock_t (long type)
  t = clock(); // start the clock

  initialize_matrices(A, B, C, ROWS, COLUMNS);

  t = clock() - t; // stop the clock
  
  double time_taken = ((double)t)/CLOCKS_PER_SEC; // convert to seconds (and long to double)
  
  printf("Initialization Time: %f sec\n",time_taken);

  /*======================================================================*/
  /*                START of Section of the code that matters!!!          */
  /*======================================================================*/

  /* Simple matrix multiplication */
  /*==============================*/
  if (true == simple)
  {

    clock_t t; // declare clock_t (long type)
    t = clock(); // start the clock

    // Transfer data from host to device memory
    hipMemcpy(D_A, A, sizeof(X_TYPE) * (ROWS * COLUMNS), hipMemcpyHostToDevice);
    hipMemcpy(D_B, B, sizeof(X_TYPE) * (ROWS * COLUMNS), hipMemcpyHostToDevice);
    //hipMemcpy(D_C, C, sizeof(X_TYPE) * (ROWS * COLUMNS), hipMemcpyHostToDevice);
    
    int block_size = 512;
    int grid_size = ((ROWS + block_size) / block_size);
    simple_matrix_multiply<<<grid_size,block_size>>>(D_A, D_B, D_C, ROWS, COLUMNS);

  // Transfer data from device to host memory
    hipMemcpy(C, D_C, sizeof(X_TYPE) * (ROWS * COLUMNS), hipMemcpyDeviceToHost);

    t = clock() - t; // stop the clock

    double time_taken = ((double)t)/CLOCKS_PER_SEC; // convert to seconds (and long to double)
    printf("GPU Compute Time: %f sec\n",time_taken);
  }


  /* OpenMP parallel matrix multiplication */
  /*=======================================*/
  if (true == openmp)
  {

    printf("This OpenMP option is not implemented in this code!!!!");
  }

  /*======================================================================*/
  /*                 END of Section of the code that matters!!!           */
  /*======================================================================*/

 // Deallocate device memory
    hipFree(D_A);
    hipFree(D_B);
    hipFree(D_C);

  // Deallocate host memory
  free(A);
  free(B);
  free(C);
}
