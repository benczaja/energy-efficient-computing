#include "hip/hip_runtime.h"
#include <stdio.h> // needed for ‘printf’ 
#include <stdlib.h> // needed for ‘RAND_MAX’ 
#include <omp.h> // needed for OpenMP 
#include <time.h> // needed for clock() and CLOCKS_PER_SEC etc
#include "helper.h" // local helper header to clean up code

#ifdef USE_DOUBLE
typedef double X_TYPE;
#else
typedef float X_TYPE;
#endif

void initialize_matrices(X_TYPE* A, X_TYPE* B, X_TYPE* C, int ROWS, int COLUMNS){
    for (int i = 0; i < ROWS * COLUMNS; i++)
        {
            A[i] = (X_TYPE) rand() / RAND_MAX ;
            B[i] = (X_TYPE) rand() / RAND_MAX ;
            C[i] = 0.0 ;
        }
}
__global__ void simple_matrix_multiply(X_TYPE** A, X_TYPE** B, X_TYPE** C, int ROWS, int COLUMNS){
    
    printf("(Simple) Matix Multiplication of 2D matricies of equal sizes (%d, %d)\n",ROWS,COLUMNS);

    for(int i=0;i<ROWS;i++)
    {
        for(int j=0;j<COLUMNS;j++)
        {
            for(int k=0;k<COLUMNS;k++)
            {
                C[i][j] += A[i][k]*B[k][j];
            }
        }
    }
}



int main( int argc, char *argv[] )  {

  int ROWS;
  int COLUMNS;
  int N;

  /* DUMB bools needed for the argument parsing logic */
  bool openmp = false;
  bool simple = true;
  bool sanity_check = false;
  
  /* VERY DUMB Argument Parsers */
  N = parse_arguments(argc, argv, &simple, &openmp, &sanity_check);
  ROWS = N;
  COLUMNS = N;


  /* declare the arrays...  better to do it as 1D arrays for CUDA */

  // First allocated them on the host (CPU)
    X_TYPE* A = (X_TYPE*)malloc((ROWS * COLUMNS) * sizeof(X_TYPE));
    X_TYPE* B = (X_TYPE*)malloc((ROWS * COLUMNS) * sizeof(X_TYPE));
    X_TYPE* C = (X_TYPE*)malloc((ROWS * COLUMNS) * sizeof(X_TYPE));

  // Then Allocate them on the GPUs
 // Allocate device memory for a
    hipMalloc((void**)&d_a, sizeof(float) * N);

    // Transfer data from host to device memory
    hipMemcpy(d_a, a, sizeof(float) * N, hipMemcpyHostToDevice);
    
  /* initialize the arrays */
  initialize_matrices(A, B, C, ROWS, COLUMNS);

  /*======================================================================*/
  /*                START of Section of the code that matters!!!          */
  /*======================================================================*/



  /* Simple matrix multiplication */
  /*==============================*/
  if (true == simple)
  {
    clock_t t; // declare clock_t (long type)
    t = clock(); // start the clock

   // Transfer data from host to device memory
    hipMemcpy(d_a, a, sizeof(float) * N, hipMemcpyHostToDevice);
    
    simple_matrix_multiply<<<1,1>>>(A, B, C, ROWS, COLUMNS);
    
    t = clock() - t; // stop the clock

    double time_taken = ((double)t)/CLOCKS_PER_SEC; // convert to seconds (and long to double)
    printf("TIME: %f sec\n",time_taken);
  }



  /* OpenMP parallel matrix multiplication */
  /*=======================================*/
  if (true == openmp)
  {

    printf("This OpenMP option is not implemented in this code!!!!");
  }

  /*======================================================================*/
  /*                 END of Section of the code that matters!!!           */
  /*======================================================================*/

  /* deallocate the arrays */
  for (int i=0; i<ROWS; i++)
  {
    free(A[i]);
    free(B[i]);
    free(C[i]);
  }
  free(A);
  free(B);
  free(C);
}
