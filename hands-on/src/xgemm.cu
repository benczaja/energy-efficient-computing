#include "hip/hip_runtime.h"
#include <stdio.h> // needed for ‘printf’ 
#include <stdlib.h> // needed for ‘RAND_MAX’ 
#include <omp.h> // needed for OpenMP 
#include <time.h> // needed for clock() and CLOCKS_PER_SEC etc
#include "helper.h" // local helper header to clean up code
#include <hiprand.h>

#ifdef USE_DOUBLE
typedef double X_TYPE;
#else
typedef float X_TYPE;
#endif

void initialize_matrices(X_TYPE* A, X_TYPE* B, X_TYPE* C, int ROWS, int COLUMNS){
    for (int i = 0; i < ROWS * COLUMNS; i++)
        {
            A[i] = (X_TYPE) rand() / RAND_MAX ;
            B[i] = (X_TYPE) rand() / RAND_MAX ;
            C[i] = 0.0 ;
        }
}


__global__ void simple_matrix_multiply(X_TYPE* D_A, X_TYPE* D_B, X_TYPE* D_C, int ROWS, int COLUMNS){
    
    int local_COLUMN = threadIdx.x + blockIdx.x * blockDim.x;
		int local_ROW = threadIdx.y + blockIdx.y * blockDim.y;
		int local_index = local_COLUMN + local_ROW * ROWS; // Right now this only works for symetric matricies
		int tmp = 0;  
    
    if(local_ROW < ROWS && local_COLUMN < COLUMNS){
			for(int k=0; k<COLUMNS; k++){
				tmp += D_A[local_ROW * ROWS + k] * D_B[k * COLUMNS + local_COLUMN];
			}
			D_C[local_index] = tmp;
		}
}


int main( int argc, char *argv[] )  {

  int ROWS;
  int COLUMNS;
  int N;

  /* DUMB bools needed for the argument parsing logic */
  bool openmp = false;
  bool simple = true;
  bool sanity_check = false;
  
  /* VERY DUMB Argument Parsers */
  N = parse_arguments(argc, argv, &simple, &openmp, &sanity_check);
  ROWS = N;
  COLUMNS = N;

  /* declare the arrays...  better to do it as 1D arrays for CUDA */

  // First allocated them on the host (CPU)
    X_TYPE* A = (X_TYPE*)malloc((ROWS * COLUMNS) * sizeof(X_TYPE));
    X_TYPE* B = (X_TYPE*)malloc((ROWS * COLUMNS) * sizeof(X_TYPE));
    X_TYPE* C = (X_TYPE*)malloc((ROWS * COLUMNS) * sizeof(X_TYPE));

  // Then Allocate them on the GPUs
  X_TYPE* D_A;
  X_TYPE* D_B;
  X_TYPE* D_C;
  hipMalloc((void**)&D_A, sizeof( X_TYPE ) * (ROWS * COLUMNS));
  hipMalloc((void**)&D_B, sizeof( X_TYPE ) * (ROWS * COLUMNS));
  hipMalloc((void**)&D_C, sizeof( X_TYPE ) * (ROWS * COLUMNS));

  // We are going to actuall 
  //size_t n = 100;
  //size_t i;
    clock_t t; // declare clock_t (long type)
    t = clock(); // start the clock

  hiprandGenerator_t gen;
  //float *devData, *hostData;
  /* Allocate n floats on host */
  //hostData = (float *)calloc(n, sizeof(float));
  /* Allocate n floats on device */
  //CUDA_CALL(hipMalloc((void **)&devData, n*sizeof(float)));
  /* Create pseudo-random number generator */
  hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
  /* Set seed */
  hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL);
  /* Generate n floats on device */
  if (sizeof(X_TYPE) == 8) {
    hiprandGenerateUniformDouble(gen, D_A, (ROWS * COLUMNS));
    hiprandGenerateUniformDouble(gen, D_B, (ROWS * COLUMNS));
  }else{
    hiprandGenerateUniform(gen, D_A, (ROWS * COLUMNS));
    hiprandGenerateUniform(gen, D_B, (ROWS * COLUMNS));    
  }
  /* Copy device memory to host */
  hipMemcpy(A, D_A, sizeof(X_TYPE) * (ROWS * COLUMNS),hipMemcpyDeviceToHost);
  hipMemcpy(B, D_B, sizeof(X_TYPE) * (ROWS * COLUMNS),hipMemcpyDeviceToHost);

    double time_taken = ((double)t)/CLOCKS_PER_SEC; // convert to seconds (and long to double)
    printf("GPU Init Time: %f sec\n",time_taken); 


  /*======================================================================*/
  /*                START of Section of the code that matters!!!          */
  /*======================================================================*/

  /* Simple matrix multiplication */
  /*==============================*/
  if (true == simple)
  {
    int block_size = 512;
    int grid_size = ((ROWS + block_size) / block_size);

    t = clock(); // start the clock

    // Transfer data from host to device memory
    hipMemcpy(D_A, A, sizeof(X_TYPE) * (ROWS * COLUMNS), hipMemcpyHostToDevice);
    hipMemcpy(D_B, B, sizeof(X_TYPE) * (ROWS * COLUMNS), hipMemcpyHostToDevice);
    
    simple_matrix_multiply<<<grid_size,block_size>>>(D_A, D_B, D_C, ROWS, COLUMNS);

   // Transfer data from device to host memory
    hipMemcpy(C, D_C, sizeof(X_TYPE) * (ROWS * COLUMNS), hipMemcpyDeviceToHost);

    t = clock() - t; // stop the clock

    time_taken = ((double)t)/CLOCKS_PER_SEC; // convert to seconds (and long to double)
    printf("GPU Compute Time: %f sec\n",time_taken);
  }


  /* OpenMP parallel matrix multiplication */
  /*=======================================*/
  if (true == openmp)
  {

    printf("This OpenMP option is not implemented in this code!!!!");
  }

  /*======================================================================*/
  /*                 END of Section of the code that matters!!!           */
  /*======================================================================*/

 // Deallocate device memory
    hipFree(D_A);
    hipFree(D_B);
    hipFree(D_C);

  // Deallocate host memory
  free(A);
  free(B);
  free(C);
}
